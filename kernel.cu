#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <chrono>
#include <ctime>
// ==== DO NOT MODIFY CODE ABOVE THIS LINE ====

#define DTYPE int
// Add any additional #include headers or helper macros needed
#include <vector>

#define NUM_STREAMS 8
#define BLOCK_SIZE 1024
#define MAX_GLOBAL_STEPS 3

// Implement your GPU device kernel(s) here (e.g., the bitonic sort kernel).

__device__ void compareExchange(DTYPE& a, DTYPE& b, bool ascending) {
  const DTYPE min_ = min(a, b);
  const DTYPE max_ = max(a, b);
  a = ascending ? min_ : max_;
  b = ascending ? max_ : min_;
}

__global__ void bitonicSortInitialShared(DTYPE* __restrict__ arr,
                                         int blockOffset) {
  __shared__ DTYPE shared[BLOCK_SIZE];
  __shared__ DTYPE dummy[2];

  const int globalIdx = threadIdx.x + (blockIdx.x + blockOffset) * BLOCK_SIZE;
  shared[threadIdx.x] = arr[globalIdx];

  __syncthreads();

  for (int i = 1; i <= __builtin_ctz(BLOCK_SIZE); i++) {
    for (int j = i - 1; j >= 0; j--) {
      const int partner = threadIdx.x ^ (1 << j);
      const bool isActive = partner > threadIdx.x;
      DTYPE& a = isActive ? shared[threadIdx.x] : dummy[0];
      DTYPE& b = isActive ? shared[partner] : dummy[1];
      const bool ascending = ((globalIdx & (1 << i)) == 0);
      compareExchange(a, b, ascending);

      __syncthreads();
    }
  }

  arr[globalIdx] = shared[threadIdx.x];
}

__global__ void bitonicSortShared(DTYPE* __restrict__ arr, int stage,
                                  int blockOffset) {
  __shared__ DTYPE shared[BLOCK_SIZE];
  __shared__ DTYPE dummy[2];

  const int globalIdx = threadIdx.x + (blockIdx.x + blockOffset) * BLOCK_SIZE;
  shared[threadIdx.x] = arr[globalIdx];

  __syncthreads();

  for (int j = __builtin_ctz(BLOCK_SIZE) - 1; j >= 0; j--) {
    const int partner = threadIdx.x ^ (1 << j);
    const bool isActive = partner > threadIdx.x;
    DTYPE& a = isActive ? shared[threadIdx.x] : dummy[0];
    DTYPE& b = isActive ? shared[partner] : dummy[1];
    const bool ascending = ((globalIdx & (1 << stage)) == 0);
    compareExchange(a, b, ascending);

    __syncthreads();
  }

  arr[globalIdx] = shared[threadIdx.x];
}

__global__ void bitonicSortGlobal(DTYPE* __restrict__ arr, int stage, int step,
                                  int blockOffset, int handleSteps) {
  const int stride = 1 << (step - handleSteps + 1);
  const int groupIdx = threadIdx.x + (blockIdx.x + blockOffset) * blockDim.x;
  const int group = groupIdx >> (step - handleSteps + 1);
  const int offset = groupIdx & (stride - 1);
  const int idx0 = (group << (step + 1)) + offset;
  const bool ascending = ((idx0 & (1 << stage)) == 0);

  if (handleSteps == 3) {
    const int idx1 = idx0 + stride;
    const int idx2 = idx1 + stride;
    const int idx3 = idx2 + stride;
    const int idx4 = idx3 + stride;
    const int idx5 = idx4 + stride;
    const int idx6 = idx5 + stride;
    const int idx7 = idx6 + stride;

    DTYPE value0 = arr[idx0];
    DTYPE value1 = arr[idx1];
    DTYPE value2 = arr[idx2];
    DTYPE value3 = arr[idx3];
    DTYPE value4 = arr[idx4];
    DTYPE value5 = arr[idx5];
    DTYPE value6 = arr[idx6];
    DTYPE value7 = arr[idx7];

    compareExchange(value0, value4, ascending);
    compareExchange(value1, value5, ascending);
    compareExchange(value2, value6, ascending);
    compareExchange(value3, value7, ascending);

    compareExchange(value0, value2, ascending);
    compareExchange(value1, value3, ascending);
    compareExchange(value4, value6, ascending);
    compareExchange(value5, value7, ascending);

    compareExchange(value0, value1, ascending);
    compareExchange(value2, value3, ascending);
    compareExchange(value4, value5, ascending);
    compareExchange(value6, value7, ascending);

    arr[idx0] = value0;
    arr[idx1] = value1;
    arr[idx2] = value2;
    arr[idx3] = value3;
    arr[idx4] = value4;
    arr[idx5] = value5;
    arr[idx6] = value6;
    arr[idx7] = value7;
  } else if (handleSteps == 2) {
    const int idx1 = idx0 + stride;
    const int idx2 = idx1 + stride;
    const int idx3 = idx2 + stride;

    DTYPE value0 = arr[idx0];
    DTYPE value1 = arr[idx1];
    DTYPE value2 = arr[idx2];
    DTYPE value3 = arr[idx3];

    compareExchange(value0, value2, ascending);
    compareExchange(value1, value3, ascending);

    compareExchange(value0, value1, ascending);
    compareExchange(value2, value3, ascending);

    arr[idx0] = value0;
    arr[idx1] = value1;
    arr[idx2] = value2;
    arr[idx3] = value3;
  } else {
    const int idx1 = idx0 + stride;

    DTYPE value0 = arr[idx0];
    DTYPE value1 = arr[idx1];

    compareExchange(value0, value1, ascending);

    arr[idx0] = value0;
    arr[idx1] = value1;
  }
}

void performBitonicSort(DTYPE* __restrict__ arrGpu,
                        std::vector<hipStream_t>& streams, int N) {
  const int logN = __builtin_ctz(N);
  const int logBlockSize = __builtin_ctz(BLOCK_SIZE);
  const int logNumStreams = __builtin_ctz(NUM_STREAMS);
  const int logStreamSize = logN - logNumStreams;

  const int totalNumBlocks = N / BLOCK_SIZE;
  const int numBlocksPerStream = totalNumBlocks / NUM_STREAMS;

  std::vector<hipEvent_t> events(NUM_STREAMS);
  for (int s = 0; s < NUM_STREAMS; s++) {
    hipEventCreate(&events[s]);
  }

  for (int s = 0; s < NUM_STREAMS; s++) {
    int blockOffset = s * numBlocksPerStream;
    bitonicSortInitialShared<<<numBlocksPerStream, BLOCK_SIZE, 0, streams[s]>>>(
        arrGpu, blockOffset);
  }

  for (int stage = logBlockSize + 1; stage <= logN; stage++) {
    int step = stage - 1;
    while (step >= logBlockSize) {
      int handleSteps = std::min(MAX_GLOBAL_STEPS, step - logBlockSize + 1);
      int blockSize = BLOCK_SIZE >> handleSteps;

      bool crossStream = (step >= logStreamSize);

      if (crossStream) {
        const int streamsPerGroup = 1 << (step + 1 - logStreamSize);
        const int blocksPerGroup = streamsPerGroup * numBlocksPerStream;
        const int numGroups = NUM_STREAMS / streamsPerGroup;

        for (int g = 0; g < numGroups; g++) {
          const int groupStart = g * streamsPerGroup;
          hipStream_t& leaderStream = streams[groupStart];

          for (int s = 0; s < streamsPerGroup; s++) {
            const int streamIdx = groupStart + s;
            hipEventRecord(events[streamIdx], streams[streamIdx]);
          }

          for (int s = 0; s < streamsPerGroup; s++) {
            const int streamIdx = groupStart + s;
            hipStreamWaitEvent(leaderStream, events[streamIdx], 0);
          }

          const int blockOffset = groupStart * numBlocksPerStream;
          bitonicSortGlobal<<<blocksPerGroup, blockSize, 0, leaderStream>>>(
              arrGpu, stage, step, blockOffset, handleSteps);

          hipEventRecord(events[groupStart], leaderStream);
          for (int s = 1; s < streamsPerGroup; s++) {
            const int followerIdx = groupStart + s;
            hipStreamWaitEvent(streams[followerIdx], events[groupStart], 0);
          }
        }
      } else {
        for (int s = 0; s < NUM_STREAMS; s++) {
          int blockOffset = s * numBlocksPerStream;
          bitonicSortGlobal<<<numBlocksPerStream, blockSize, 0, streams[s]>>>(
              arrGpu, stage, step, blockOffset, handleSteps);
        }
      }

      step -= handleSteps;
    }

    for (int s = 0; s < NUM_STREAMS; s++) {
      int blockOffset = s * numBlocksPerStream;
      bitonicSortShared<<<numBlocksPerStream, BLOCK_SIZE, 0, streams[s]>>>(
          arrGpu, stage, blockOffset);
    }
  }

  for (int s = 0; s < NUM_STREAMS; s++) {
    hipEventDestroy(events[s]);
  }
}

/* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
int main(int argc, char* argv[]) {
  if (argc < 2) {
    printf("Usage: %s <array_size>\n", argv[0]);
    return 1;
  }

  int size = atoi(argv[1]);

  srand(time(NULL));

  DTYPE* arrCpu = (DTYPE*)malloc(size * sizeof(DTYPE));

  for (int i = 0; i < size; i++) {
    arrCpu[i] = rand() % 1000;
  }

  float gpuTime, h2dTime, d2hTime, cpuTime = 0;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  /* ==== DO NOT MODIFY CODE ABOVE THIS LINE ==== */

  std::vector<hipStream_t> streams(NUM_STREAMS);
  for (int i = 0; i < NUM_STREAMS; i++) {
    hipStreamCreate(&streams[i]);
  }

  hipStream_t paddingStream;
  hipStreamCreate(&paddingStream);
  hipEvent_t paddingCompleteEvent;
  hipEventCreate(&paddingCompleteEvent);

  DTYPE* arrGpu;
  int N = BLOCK_SIZE * NUM_STREAMS;
  while (N < size) {
    N <<= 1;
  }
  hipMalloc((void**)&arrGpu, N * sizeof(DTYPE));

  const int paddingLength = N - size;
  if (paddingLength > 0) {
    hipMemsetAsync(arrGpu + size, 0, paddingLength * sizeof(DTYPE),
                    paddingStream);
    hipEventRecord(paddingCompleteEvent, paddingStream);
  }

  // Transfer data (arrCpu) to device
  const int chunkSize = N / NUM_STREAMS;
  int copied = 0;
  for (int i = 0; i < NUM_STREAMS && copied < size; i++) {
    const int copySize = std::min(chunkSize, size - copied);
    hipHostRegister(arrCpu + copied, copySize * sizeof(DTYPE),
                     hipHostRegisterDefault);
    hipMemcpyAsync(arrGpu + copied, arrCpu + copied, copySize * sizeof(DTYPE),
                    hipMemcpyHostToDevice, streams[i]);
    copied += copySize;
  }

  if (paddingLength > 0) {
    for (int i = NUM_STREAMS / 2; i < NUM_STREAMS; i++) {
      hipStreamWaitEvent(streams[i], paddingCompleteEvent, 0);
    }
  }

  /* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&h2dTime, start, stop);

  hipEventRecord(start);

  /* ==== DO NOT MODIFY CODE ABOVE THIS LINE ==== */

  // Perform bitonic sort on GPU
  performBitonicSort(arrGpu, streams, N);

  DTYPE* arrSortedGpu = (DTYPE*)malloc(size * sizeof(DTYPE));
  hipHostRegister(arrSortedGpu, size * sizeof(DTYPE), hipHostRegisterDefault);

  /* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpuTime, start, stop);

  hipEventRecord(start);

  /* ==== DO NOT MODIFY CODE ABOVE THIS LINE ==== */

  // Transfer sorted data back to host (copied to arrSortedGpu)
  copied = 0;
  for (int i = NUM_STREAMS - 1; i >= 0 && copied < size; i--) {
    const int copySize = std::min(chunkSize, size - copied);
    const int destOffset = size - copied - copySize;
    const int srcOffset = N - copied - copySize;
    hipMemcpyAsync(arrSortedGpu + destOffset, arrGpu + srcOffset,
                    copySize * sizeof(DTYPE), hipMemcpyDeviceToHost,
                    streams[i]);
    copied += copySize;
  }

  for (int i = 0; i < NUM_STREAMS; i++) {
    hipStreamDestroy(streams[i]);
  }
  hipStreamDestroy(paddingStream);
  hipEventDestroy(paddingCompleteEvent);

  // Not a requirement for the project
  // (https://edstem.org/us/courses/81715/discussion/6897777?comment=16332533)
  // cudaHostUnregister(arrCpu);
  // cudaHostUnregister(arrSortedGpu);

  hipFreeAsync(arrGpu, 0);

  /* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&d2hTime, start, stop);

  auto startTime = std::chrono::high_resolution_clock::now();

  // CPU sort for performance comparison
  std::sort(arrCpu, arrCpu + size);

  auto endTime = std::chrono::high_resolution_clock::now();
  cpuTime =
      std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime)
          .count();
  cpuTime = cpuTime / 1000;

  int match = 1;
  for (int i = 0; i < size; i++) {
    if (arrSortedGpu[i] != arrCpu[i]) {
      match = 0;
      break;
    }
  }

  free(arrCpu);
  free(arrSortedGpu);

  if (match)
    printf("\033[1;32mFUNCTIONAL SUCCESS\n\033[0m");
  else {
    printf("\033[1;31mFUNCTIONCAL FAIL\n\033[0m");
    return 0;
  }

  printf("\033[1;34mArray size         :\033[0m %d\n", size);
  printf("\033[1;34mCPU Sort Time (ms) :\033[0m %f\n", cpuTime);
  float gpuTotalTime = h2dTime + gpuTime + d2hTime;
  int speedup = (gpuTotalTime > cpuTime) ? (gpuTotalTime / cpuTime)
                                         : (cpuTime / gpuTotalTime);
  float meps = size / (gpuTotalTime * 0.001) / 1e6;
  printf("\033[1;34mGPU Sort Time (ms) :\033[0m %f\n", gpuTotalTime);
  printf(
      "\033[1;34mGPU Sort Speed     :\033[0m %f million elements per second\n",
      meps);
  if (gpuTotalTime < cpuTime) {
    printf("\033[1;32mPERF PASSING\n\033[0m");
    printf(
        "\033[1;34mGPU Sort is \033[1;32m %dx \033[1;34mfaster than CPU "
        "!!!\033[0m\n",
        speedup);
    printf("\033[1;34mH2D Transfer Time (ms):\033[0m %f\n", h2dTime);
    printf("\033[1;34mKernel Time (ms)      :\033[0m %f\n", gpuTime);
    printf("\033[1;34mD2H Transfer Time (ms):\033[0m %f\n", d2hTime);
  } else {
    printf("\033[1;31mPERF FAILING\n\033[0m");
    printf(
        "\033[1;34mGPU Sort is \033[1;31m%dx \033[1;34mslower than CPU, "
        "optimize further!\n",
        speedup);
    printf("\033[1;34mH2D Transfer Time (ms):\033[0m %f\n", h2dTime);
    printf("\033[1;34mKernel Time (ms)      :\033[0m %f\n", gpuTime);
    printf("\033[1;34mD2H Transfer Time (ms):\033[0m %f\n", d2hTime);
    return 0;
  }

  return 0;
}