#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <chrono>
#include <ctime>
// ==== DO NOT MODIFY CODE ABOVE THIS LINE ====

#define DTYPE int
// Add any additional #include headers or helper macros needed
#define NUM_STREAMS 4
#define COMPARATOR_WIDTH 4
#define BLOCK_SIZE 1024
#define SHARED_SIZE (BLOCK_SIZE * COMPARATOR_WIDTH)

// Implement your GPU device kernel(s) here (e.g., the bitonic sort kernel).

// Branchless compare-exchange
__device__ void compareExchange(DTYPE& a, DTYPE& b, bool ascending) {
  DTYPE lessMask = -((a - b) < 0);
  DTYPE min = (lessMask & a) | (~lessMask & b);
  DTYPE max = (lessMask & b) | (~lessMask & a);

  DTYPE ascMask = -(ascending);
  a = (ascMask & min) | (~ascMask & max);
  b = (ascMask & max) | (~ascMask & min);
}

// Hand-coded bitonic sorting network for 2 elements
__device__ void bitonicSortingNetwork2(DTYPE* a, DTYPE* b, bool ascending) {
  compareExchange(a[0], b[0], ascending);
}

// Hand-coded bitonic sorting network for 4 elements
__device__ void bitonicSortingNetwork4(DTYPE* a, DTYPE* b, bool ascending) {
  bitonicSortingNetwork2(&a[0], &a[1], ascending);
  bitonicSortingNetwork2(&b[0], &b[1], !ascending);

  compareExchange(a[0], b[0], ascending);
  compareExchange(a[1], b[1], ascending);

  bitonicSortingNetwork2(&a[0], &a[1], ascending);
  bitonicSortingNetwork2(&b[0], &b[1], ascending);
}

// Hand-coded bitonic sorting network for 8 elements
__device__ void bitonicSortingNetwork8(DTYPE* a, DTYPE* b, bool ascending) {
  bitonicSortingNetwork4(&a[0], &a[2], ascending);
  bitonicSortingNetwork4(&b[0], &b[2], !ascending);

  compareExchange(a[0], b[0], ascending);
  compareExchange(a[1], b[1], ascending);
  compareExchange(a[2], b[2], ascending);
  compareExchange(a[3], b[3], ascending);

  bitonicSortingNetwork4(&a[0], &a[2], ascending);
  bitonicSortingNetwork4(&b[0], &b[2], ascending);
}

// Compare-exchange for COMPARATOR_WIDTH elements
__device__ void compareExchangeBlock(DTYPE* a, DTYPE* b, bool ascending) {
#if COMPARATOR_WIDTH == 1
  bitonicSortingNetwork2(a, b, ascending);
#elif COMPARATOR_WIDTH == 2
  bitonicSortingNetwork4(a, b, ascending);
#elif COMPARATOR_WIDTH == 4
  bitonicSortingNetwork8(a, b, ascending);
#else
#error "Provided COMPARATOR_WIDTH not supported."
#endif
}

__global__ void bitonicSortInitialShared(DTYPE* arr) {
  __shared__ DTYPE shared[SHARED_SIZE];

  int k = threadIdx.x;
  int sharedIdx = k * COMPARATOR_WIDTH;
  int globalIdx = blockIdx.x * SHARED_SIZE + sharedIdx;

  for (int i = 0; i < COMPARATOR_WIDTH; i++) {
    shared[sharedIdx + i] = arr[globalIdx + i];
  }

  __syncthreads();

  for (int i = 1; i <= __builtin_ctz(BLOCK_SIZE); i++) {
    for (int j = i - 1; j >= 0; j--) {
      int partner = k ^ (1 << j);
      if (partner > k) {
        bool ascending = ((k & (1 << i)) == 0);
        compareExchangeBlock(&shared[sharedIdx],
                             &shared[partner * COMPARATOR_WIDTH], ascending);
      }

      __syncthreads();
    }
  }

  for (int i = 0; i < COMPARATOR_WIDTH; i++) {
    arr[globalIdx + i] = shared[sharedIdx + i];
  }
}

__global__ void bitonicSortShared(DTYPE* arr, int stage) {
  __shared__ DTYPE shared[SHARED_SIZE];

  int k = threadIdx.x;
  int sharedIdx = k * COMPARATOR_WIDTH;
  int globalIdx = blockIdx.x * SHARED_SIZE + sharedIdx;

  for (int i = 0; i < COMPARATOR_WIDTH; i++) {
    shared[sharedIdx + i] = arr[globalIdx + i];
  }

  __syncthreads();

  for (int j = __builtin_ctz(BLOCK_SIZE) - 1; j >= 0; j--) {
    int partner = k ^ (1 << j);
    if (partner > k) {
      bool ascending = ((k & (1 << stage)) == 0);
      compareExchangeBlock(&shared[sharedIdx],
                           &shared[partner * COMPARATOR_WIDTH], ascending);
    }

    __syncthreads();
  }

  for (int i = 0; i < COMPARATOR_WIDTH; i++) {
    arr[globalIdx + i] = shared[sharedIdx + i];
  }
}

__global__ void bitonicSortGlobal(DTYPE* arr, int stage, int step) {
  int k = threadIdx.x + blockIdx.x * blockDim.x;
  int partner = k ^ (1 << step);

  if (partner > k) {
    bool ascending = ((k & (1 << stage)) == 0);
    compareExchangeBlock(&arr[k * COMPARATOR_WIDTH],
                         &arr[partner * COMPARATOR_WIDTH], ascending);
  }
}

void performBitonicSort(DTYPE* arrGpu, std::vector<hipStream_t>& streams,
                        int N, int logN) {
  // TODO: bitonicSort<<<grid, block>>>(arrGpu);

  int i = 1;

  // Invoke bitonicSortInitialShared and increment i appropriately

  while (i <= logN) {
    int j = i - 1;
    while (j >= 0) {
      // Figure out whether to call bitonicSortGlobal or bitonicSortShared
      // If bitonicSortGlobal, decrement j by (at most) log(COMPARATOR_WIDTH)
      // If bitonicSortShared, break

      // j--;
    }
    i++;
  }
}

/* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
int main(int argc, char* argv[]) {
  if (argc < 2) {
    printf("Usage: %s <array_size>\n", argv[0]);
    return 1;
  }

  int size = atoi(argv[1]);

  srand(time(NULL));

  DTYPE* arrCpu = (DTYPE*)malloc(size * sizeof(DTYPE));

  for (int i = 0; i < size; i++) {
    arrCpu[i] = rand() % 1000;
  }

  float gpuTime, h2dTime, d2hTime, cpuTime = 0;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  /* ==== DO NOT MODIFY CODE ABOVE THIS LINE ==== */

  hipHostRegister(arrCpu, size * sizeof(DTYPE), hipHostRegisterDefault);

  std::vector<hipStream_t> streams(NUM_STREAMS);
  for (int i = 0; i < NUM_STREAMS; i++) {
    hipStreamCreate(&streams[i]);
  }

  // arCpu contains the input random array
  // arrSortedGpu should contain the sorted array copied from GPU to CPU
  DTYPE* arrSortedGpu = (DTYPE*)malloc(size * sizeof(DTYPE));

  DTYPE* arrGpu;

  int N = 1;
  int logN = 0;
  while (N < size) {
    N <<= 1;
    logN++;
  }
  hipMalloc((void**)&arrGpu, N * sizeof(DTYPE));
  cudaMemsetD32Async(arrGpu + size, INT_MAX, N - size, 0);

  // Transfer data (arrCpu) to device
  int chunkSize = N / NUM_STREAMS;
  int copied = 0;
  for (int i = 0; i < NUM_STREAMS && copied < size; i++) {
    int copySize = std::min(chunkSize, size - copied);
    hipMemcpyAsync(arrGpu + copied, arrCpu + copied, copySize * sizeof(DTYPE),
                    hipMemcpyHostToDevice, streams[i]);
    copied += chunkSize;
  }

  /* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&h2dTime, start, stop);

  hipEventRecord(start);

  /* ==== DO NOT MODIFY CODE ABOVE THIS LINE ==== */

  // Perform bitonic sort on GPU
  performBitonicSort(arrGpu, streams, N, logN);

  hipHostRegister(arrSortedGpu, size * sizeof(DTYPE), hipHostRegisterDefault);

  /* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpuTime, start, stop);

  hipEventRecord(start);

  /* ==== DO NOT MODIFY CODE ABOVE THIS LINE ==== */

  // Transfer sorted data back to host (copied to arrSortedGpu)
  copied = 0;
  for (int i = 0; i < NUM_STREAMS && copied < size; i++) {
    int copySize = std::min(chunkSize, size - copied);
    hipMemcpyAsync(arrSortedGpu + copied, arrGpu + copied,
                    copySize * sizeof(DTYPE), hipMemcpyDeviceToHost,
                    streams[i]);
    copied += chunkSize;
  }

  for (int i = 0; i < NUM_STREAMS; i++) {
    hipStreamDestroy(&streams[i]);
  }

  // Not a requirement for the project
  // (https://edstem.org/us/courses/81715/discussion/6897777?comment=16332533)
  // hipHostUnregister(arrCpu);
  // hipHostUnregister(arrSortedGpu);

  hipFree(arrGpu);

  /* ==== DO NOT MODIFY CODE BELOW THIS LINE ==== */
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&d2hTime, start, stop);

  auto startTime = std::chrono::high_resolution_clock::now();

  // CPU sort for performance comparison
  std::sort(arrCpu, arrCpu + size);

  auto endTime = std::chrono::high_resolution_clock::now();
  cpuTime =
      std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime)
          .count();
  cpuTime = cpuTime / 1000;

  int match = 1;
  for (int i = 0; i < size; i++) {
    if (arrSortedGpu[i] != arrCpu[i]) {
      match = 0;
      break;
    }
  }

  free(arrCpu);
  free(arrSortedGpu);

  if (match)
    printf("\033[1;32mFUNCTIONAL SUCCESS\n\033[0m");
  else {
    printf("\033[1;31mFUNCTIONCAL FAIL\n\033[0m");
    return 0;
  }

  printf("\033[1;34mArray size         :\033[0m %d\n", size);
  printf("\033[1;34mCPU Sort Time (ms) :\033[0m %f\n", cpuTime);
  float gpuTotalTime = h2dTime + gpuTime + d2hTime;
  int speedup = (gpuTotalTime > cpuTime) ? (gpuTotalTime / cpuTime)
                                         : (cpuTime / gpuTotalTime);
  float meps = size / (gpuTotalTime * 0.001) / 1e6;
  printf("\033[1;34mGPU Sort Time (ms) :\033[0m %f\n", gpuTotalTime);
  printf(
      "\033[1;34mGPU Sort Speed     :\033[0m %f million elements per second\n",
      meps);
  if (gpuTotalTime < cpuTime) {
    printf("\033[1;32mPERF PASSING\n\033[0m");
    printf(
        "\033[1;34mGPU Sort is \033[1;32m %dx \033[1;34mfaster than CPU "
        "!!!\033[0m\n",
        speedup);
    printf("\033[1;34mH2D Transfer Time (ms):\033[0m %f\n", h2dTime);
    printf("\033[1;34mKernel Time (ms)      :\033[0m %f\n", gpuTime);
    printf("\033[1;34mD2H Transfer Time (ms):\033[0m %f\n", d2hTime);
  } else {
    printf("\033[1;31mPERF FAILING\n\033[0m");
    printf(
        "\033[1;34mGPU Sort is \033[1;31m%dx \033[1;34mslower than CPU, "
        "optimize further!\n",
        speedup);
    printf("\033[1;34mH2D Transfer Time (ms):\033[0m %f\n", h2dTime);
    printf("\033[1;34mKernel Time (ms)      :\033[0m %f\n", gpuTime);
    printf("\033[1;34mD2H Transfer Time (ms):\033[0m %f\n", d2hTime);
    return 0;
  }

  return 0;
}
